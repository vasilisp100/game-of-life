#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>
#include <assert.h>
#include <time.h>

/*COLOR*/
#define RED "\x1B[31m"
#define BLUE "\x1B[34m"

/*BOLD-COLOR*/
#define B_RED "\x1B[1m\x1B[31m"
#define B_GREEN "\x1B[1m\x1B[32m"

/*RESET COLOR*/
#define RESET  "\x1B[0m"

#define N 16
#define M 4
#define FILE_NAME "/home/msi/projects/CLionProjects/game-of-life/cuda/test-files/64x64.txt"
#define STEPS 1

int **allocate2DArray(int rows, int columns) {
    int **block;
    int i;
    block = (int **) malloc(rows * sizeof(int *));
    block[0] = (int *) malloc(rows * columns * sizeof(int));
    for (i = 1; i < rows; i++) {
        block[i] = &(block[0][i * rows]);
    }
    memset(block[0], 0, rows * columns * sizeof(int));
    return block;
}

void free2DArray(int **block) {
    free(block[0]);
    free(block);
}

void print_array(int **array, bool split, bool internals, int rowDim, int colDim, int localRowDim, int localColDim) {
    printf("\n");
    for (int i = 0; i < rowDim; i++) {
        for (int j = 0; j < colDim; j++) {
            if ((rowDim != localRowDim && colDim != localColDim)) {
//                printf("%s %c ", array[i][j] == '1' ? RED"\u2B1B" RESET : "\u2B1C",
//                       (split && (j + 1) % localColDim == 0) ? ' ' : '\0');
                if (array[i][j]) {
                    printf(RED"%5.4d%c" RESET, array[i][j], (split && (j + 1) % localColDim == 0) ? ' ' : '\0');
                } else {
                    printf("%5.4d%c" RESET, array[i][j], (split && (j + 1) % localColDim == 0) ? ' ' : '\0');
                }
            } else {
                if ((i == 0 || i == rowDim - 1) || (j == 0 || j == colDim - 1)) {
//                    printf("%s %c ", array[i][j] == '1' ? B_GREEN"\u2B1B" RESET : "\u2B1C",
//                           (split && (j + 1) % localColDim == 0) ? ' ' : '\0');
                    if (array[i][j]) {
                        printf(RED"%5.4d%c" RESET, array[i][j], (split && (j + 1) % localColDim == 0) ? ' ' : '\0');
                    } else {
                        printf("%5.4d%c" RESET, array[i][j], (split && (j + 1) % localColDim == 0) ? ' ' : '\0');
                    }
                } else if (internals && ((i == 1 || i == rowDim - 2) || (j == 1 || j == colDim - 2))) {
//                    printf("%s %c ", array[i][j] == '1' ? BLUE"\u2B1B" RESET : "\u2B1C",
//                           (split && (j + 1) % localColDim == 0) ? ' ' : '\0');
                    if (array[i][j]) {
                        printf(RED"%5.4d%c" RESET, array[i][j], (split && (j + 1) % localColDim == 0) ? ' ' : '\0');
                    } else {
                        printf("%5.4d%c" RESET, array[i][j], (split && (j + 1) % localColDim == 0) ? ' ' : '\0');
                    }
                } else {
//                    printf("%s %c ", array[i][j] == '1' ? RED"\u2B1B" RESET : "\u2B1C",
//                           (split && (j + 1) % localColDim == 0) ? ' ' : '\0');
                    if (array[i][j]) {
                        printf(RED"%5.4d%c" RESET, array[i][j], (split && (j + 1) % localColDim == 0) ? ' ' : '\0');
                    } else {
                        printf("%5.4d%c" RESET, array[i][j], (split && (j + 1) % localColDim == 0) ? ' ' : '\0');
                    }
                }
            }
        }
        printf("\n%c", (split && (i + 1) % localRowDim == 0) ? '\n' : '\0');
    }
    printf("\n");
}

// Device code
__global__ void kernel(int *old, int *current, int *changes) {
    __shared__ int local[M + 2][M + 2];
    int sum = 0;
    unsigned int local_row = threadIdx.x;
    unsigned int local_col = threadIdx.y;
    unsigned int local_thread_id = local_col + local_row * M;

    unsigned int ix = blockIdx.x * (blockDim.x) + threadIdx.x;
    unsigned int iy = blockIdx.y * (blockDim.y) + threadIdx.y;
    unsigned int idx = ix * N + iy;

    // Todo: get rid of this
    old[idx] = idx;

    // Initialize 'local' shared array
    local[local_row + 1][local_col + 1] = old[idx];

    // Initialize neighbors
    if (blockIdx.x > 0 && blockIdx.x < gridDim.x - 1 && blockIdx.y > 0 && blockIdx.y < gridDim.y - 1) {
        //up
        if (local_row == 0) {
            local[local_row][local_col + 1] = old[idx - N];
        }
        //down
        if (local_row == blockDim.x - 1) {
            local[local_row + 2][local_col + 1] = old[idx + N];
        }
        //left
        if (local_col == 0) {
            local[local_row + 1][local_col] = old[idx - 1];
        }
        //right
        if (local_col == blockDim.y - 1) {
            local[local_row + 1][local_col + 2] = old[idx + 1];
        }
        //up left
        if (local_col == 0 && local_row == 0) {
            local[local_row][local_col] = old[idx - N - 1];
        }
        //up right
        if (local_col == blockDim.y - 1 && local_row == 0) {
            local[local_row][local_col + 2] = old[idx - N + 1];
        }
        //down left
        if (local_col == 0 && local_row == blockDim.y - 1) {
            local[local_row + 2][local_col] = old[idx + N - 1];
        }
        //down right
        if (local_col == blockDim.y - 1 && local_row == blockDim.x - 1) {
            local[local_row + 2][local_col + 2] = old[idx + N + 1];
        }
    } else {
        if (blockIdx.x == 0) {
            //up
            if (local_row == 0) {
                local[local_row][local_col + 1] = old[idx + (N - 1) * N];
            }
            //down
            if (local_row == blockDim.x - 1) {
                local[local_row + 2][local_col + 1] = old[idx + N];
            }
            //left
            if (local_col == 0) {
                //full aristera
                if (blockIdx.y == 0) {
                    local[local_row + 1][local_col] = old[idx + N - 1];
                } else {
                    local[local_row + 1][local_col] = old[idx - 1];
                }
            }
            //right
            if (local_col == blockDim.y - 1) {

                if (blockIdx.y != gridDim.y - 1) {

                    local[local_row + 1][local_col + 2] = old[idx + 1];
                }
                    //full deksia
                else {
                    local[local_row + 1][local_col + 2] = old[idx - N + 1];
                }
            }
            //up left
            if (local_col == 0 && local_row == 0) {
                //terma aristera block
                if (blockIdx.y == 0) {
                    local[local_row][local_col] = old[idx + N * N - 1];
                } else {

                    local[local_row][local_col] = old[idx + (N - 1) * N - 1];
                }
            }
            //up right
            if (local_row == 0 && local_col == blockDim.y - 1) {
                if (blockIdx.y != gridDim.y - 1) {
                    local[local_row][local_col + 2] = old[idx + (N - 1) * N + 1];
                }
                    //terma deksia
                else {
                    local[local_row][local_col + 2] = old[idx + (N - 1) * N - N + 1];
                }
            }
            //down left
            if (local_row == blockDim.x - 1 && local_col == 0) {
                //terma aristera block
                if (blockIdx.y == 0) {
                    local[local_row + 2][local_col] = old[idx + 2 * N - 1];
                } else {
                    local[local_row + 2][local_col] = old[idx + 2 * N - 1 - N];
                }
            }
            //down right
            if (local_row == blockDim.x - 1 && local_col == blockDim.y - 1) {
                if (blockIdx.y != gridDim.y - 1) {
                    local[local_row + 2][local_col + 2] = old[idx + 1 * N + 1];
                }
                    //terma deksia
                else {
                    local[local_row + 2][local_col + 2] = old[idx + N + 1];
                    // printf("local_row+3: %d, local_col+3: %d\n", local_row+3, local_col+3);
                    // printf("old[idx + N +2]: %5.4d\n", old[idx + N +2]);
                }
            }
        }

        if (blockIdx.x == gridDim.x - 1) {
            //up
            if (local_row == 0) {
                //idio me apo panw
                local[local_row][local_col + 1] = old[idx - N];
            }
            //down
            if (local_row == blockDim.x - 1) {
                local[local_row + 2][local_col + 1] = old[idx - N * (N - 1)];
            }
            //left
            if (local_col == 0) {
                //full aristera
                if (blockIdx.y == 0) {
                    local[local_row + 1][local_col] = old[idx + N - 1];
                } else {
                    local[local_row + 1][local_col] = old[idx - 1];
                }
            }
            //right
            if (local_col == blockDim.y - 1) {

                if (blockIdx.y != gridDim.y - 1) {

                    local[local_row + 1][local_col + 2] = old[idx + 1];
                }
                    //full deksia
                else {
                    local[local_row + 1][local_col + 2] = old[idx - N + 1];

                }
            }
            //up left
            if (local_col == 0 && local_row == 0) {
                //terma aristera block
                if (blockIdx.y == 0) {
                    local[local_row][local_col] = old[idx - 1];
                } else {
                    local[local_row][local_col] = old[idx - N - 1];
                }
            }
            //up right
            if (local_row == 0 && local_col == blockDim.y - 1) {
                if (blockIdx.y != gridDim.y - 1) {
                    local[local_row][local_col + 2] = old[idx - N + 1];

                } else { //terma deksia
                    local[local_row][local_col + 2] = old[idx - 2 * N + 1];
                }
            }
            //down left
            if (local_row == blockDim.x - 1 && local_col == 0) {
                //terma aristera block
                if (blockIdx.y == 0) {
                    local[local_row + 2][local_col] = old[idx - (N - 1) * (N - 1)];
                } else {
                    local[local_row + 2][local_col] = old[idx - N * (N - 1) - 1];
                }
            }
            //down right
            if (local_row == blockDim.x - 1 && local_col == blockDim.y - 1) {
                if (blockIdx.y != gridDim.y - 1) {

                    local[local_row + 2][local_col + 2] = old[idx - (N - 1) * N + 1];
                } else if (blockIdx.y == gridDim.y - 1) {
                    local[local_row + 2][local_col + 2] = old[idx - (N - 1) * N + 1 - N];
                }

            }
        }

        if (blockIdx.x > 0 && blockIdx.x < gridDim.x - 1 && blockIdx.y == 0) {
            //up
            if (local_row == 0) {
                local[local_row][local_col + 1] = old[idx - N];
            }
            //down
            if (local_row == blockDim.x - 1) {
                local[local_row + 2][local_col + 1] = old[idx + N];
            }
            //right
            if (local_col == blockDim.y - 1) {
                local[local_row + 1][local_col + 2] = old[idx + 1];
            }
            //left
            if (local_col == 0) {
                local[local_row + 1][local_col] = old[idx + N - 1];
            }
            //up right
            if (local_col == blockDim.y - 1 && local_row == 0) {
                local[local_row][local_col + 2] = old[idx - N + 1];
            }
            //down right
            if (local_col == blockDim.y - 1 && local_row == blockDim.x - 1) {
                local[local_row + 2][local_col + 2] = old[idx + N + 1];
            }
            //up left
            if (local_col == 0 && local_row == 0) {
                //terma aristera block
                if (blockIdx.y == 0) {
                    local[local_row][local_col] = old[idx - 1];
                }

            }
            //down left
            if (local_row == blockDim.x - 1 && local_col == 0) {
                //terma aristera block
                if (blockIdx.y == 0) {
                    local[local_row + 2][local_col] = old[idx + 2 * N - 1];
                }
            }
        }

        if (blockIdx.x > 0 && blockIdx.x < gridDim.x - 1 && blockIdx.y == gridDim.y - 1) {
            //up
            if (local_row == 0) {
                local[local_row][local_col + 1] = old[idx - N];
            }
            //down
            if (local_row == blockDim.x - 1) {
                local[local_row + 2][local_col + 1] = old[idx + N];
            }
            //left
            if (local_col == 0) {
                local[local_row + 1][local_col] = old[idx - 1];
            }
            //up left
            if (local_col == 0 && local_row == 0) {
                local[local_row][local_col] = old[idx - N - 1];
            }
            //down left
            if (local_col == 0 && local_row == blockDim.y - 1) {
                local[local_row + 2][local_col] = old[idx + N - 1];
            }
            //right
            if (local_col == blockDim.y - 1) {
                local[local_row + 1][local_col + 2] = old[idx - N + 1];
            }
            //up right
            if (local_row == 0 && local_col == blockDim.y - 1) {
                local[local_row][local_col + 2] = old[idx - 2 * N + 1];
            }
            //down right
            if (local_row == blockDim.x - 1 && local_col == blockDim.y - 1) {
                local[local_row + 2][local_col + 2] = old[idx + 1];
            }
        }
    }

    __syncthreads();

    // Calculate cells
    sum = (local[local_row][local_col] - 48) +
          (local[local_row][local_col + 1] - 48) +
          (local[local_row][local_col + 2] - 48) +
          (local[local_row + 1][local_col] - 48) +
          (local[local_row + 1][local_col + 2] - 48) +
          (local[local_row + 2][local_col] - 48) +
          (local[local_row + 2][local_col + 1] - 48) +
          (local[local_row + 2][local_col + 2] - 48);

    // Is alive
    if ((local[local_row][local_col]) == '1') {
        if (sum <= 1 || sum >= 4) {
            current[idx] = '0';
            (*changes)++;
        } else {
            current[idx] = '1';
        }
    } else if (sum == 3) {
        current[idx] = '1';
        (*changes)++;
    } else {
        current[idx] = '0';
    }

}

// Host code
int main() {
    int **host_array = nullptr, *device_old = nullptr, *device_current = nullptr, *temp = nullptr;;
    int *device_changes = nullptr;
    int i = 0, fd = 0, changes = 0;
    double time_spent = 0.0;
    clock_t begin, end;

    // Threads (2D) per block
    dim3 m(M, M);

    // Blocks (2D grid)
    dim3 n((unsigned int) ((N + (float) M - 1) / (float) M), (unsigned int) ((N + (float) M - 1) / (float) M));

    assert(N * N == M * M * (n.x * n.y));

    // Array allocations
    host_array = allocate2DArray(N, N);

//    // Read file
//    if ((fd = open(FILE_NAME, O_RDONLY)) < 0) {
//        fprintf(stderr, "Could not open file \"%s\"\n", FILE_NAME);
//        return -1;
//    }
//    i = 1;
//    while (read(fd, &host_array[i++][1], N));
//    close(fd);

    printf("host_array before:\n");
    print_array(host_array, true, true, N, N, N, N);

    // Allocate 2D 'old' array on device
    hipMalloc((void **) &device_old, N * N * sizeof(int));

    // Allocate 'changes' int variable on device
    hipMalloc((void **) &device_changes, sizeof(int));

    // Initialize 'changes' int variable on device
    hipMemset(device_changes, 0, 1);

    // Copy 2D 'old' array on device
    hipMemcpy(device_old, host_array[0], N * N * sizeof(int), hipMemcpyHostToDevice);

    // Allocate 2D 'current' array on device
    hipMalloc((void **) &device_current, N * N * sizeof(int));

    // Initialize 2D 'current' array on device
    hipMemset(device_current, '0', N * N * sizeof(int));

    begin = clock();

    // Computations
    for (i = 0; i < STEPS; i++) {
        // Call device function
        kernel<<<n, m>>>(device_old, device_current, device_changes);

        // Copy 2D 'device_current' array from device to host
        hipMemcpy(host_array[0], device_old, sizeof(int) * N * N, hipMemcpyDeviceToHost);

        // Copy 'changes' variable from device to host
        hipMemcpy(&changes, device_changes, sizeof(int), hipMemcpyDeviceToHost);

        printf("host_array on step %d:\n", i);
        print_array(host_array, true, true, N, N, N, N);
        printf("Changes: %d:\n", changes);


        // Swap 'device_old' and 'device_current' arrays
        temp = device_old;
        device_old = device_current;
        device_current = temp;
    }

    end = clock();

    time_spent = (double) (end - begin) / CLOCKS_PER_SEC;

    printf("time_spent=%f\n", time_spent);

    // Free memory
    hipFree(device_old);
    hipFree(device_current);
    free2DArray(host_array);
    return 0;
}
